#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/ceil_div.h>

#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAException.h> // For C10_CUDA_CHECK
#include <c10/cuda/CUDACachingAllocator.h>

// #include <THC/THC.h>
// #include <THC/THCDeviceUtils.cuh>

#include <vector>
#include <iostream>
#include <cstring>

int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS + 1e-10f);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    // const int col_blocks = THCCeilDiv(n_boxes, threadsPerBlock);
    const int col_blocks = (n_boxes + threadsPerBlock - 1) / threadsPerBlock;
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

// boxes is a N x 5 tensor
at::Tensor nms_cuda(const at::Tensor boxes, float nms_overlap_thresh) {
  using scalar_t = float;
  TORCH_CHECK(boxes.is_cuda(), "boxes must be a CUDA tensor");
  TORCH_CHECK(boxes.dim() == 2 && boxes.size(1) == 5, "boxes must be Nx5");
  
  // AT_ASSERTM(boxes.device().is_cuda(), "boxes must be a CUDA tensor");
  auto scores = boxes.select(1, 4);
  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t).contiguous();

  // int boxes_num = boxes.size(0);
  const int boxes_num = static_cast<int>(boxes_sorted.size(0));

  if (boxes_num == 0) {
    return at::empty({0}, boxes.options().dtype(at::kLong).device(at::kCPU));
  }

  c10::cuda::CUDAGuard device_guard(boxes.device());
  auto stream = c10::cuda::getCurrentCUDAStream();
  
  // const int col_blocks = THCCeilDiv(boxes_num, threadsPerBlock);
  const int col_blocks = static_cast<int>(at::ceil_div(static_cast<int64_t>(boxes_num),static_cast<int64_t>(threadsPerBlock)));

  scalar_t* boxes_dev = boxes_sorted.data_ptr<scalar_t>();

  // THCState *state = at::globalContext().lazyInitCUDA(); // TODO replace with getTHCState

  // unsigned long long* mask_dev = NULL;
  // //THCudaCheck(THCudaMalloc(state, (void**) &mask_dev,
  // //                      boxes_num * col_blocks * sizeof(unsigned long long)));

  // mask_dev = (unsigned long long*) THCudaMalloc(state, boxes_num * col_blocks * sizeof(unsigned long long));

  unsigned long long* mask_dev = nullptr;
  const size_t mask_bytes = static_cast<size_t>(boxes_num) * static_cast<size_t>(col_blocks) * sizeof(unsigned long long);
  mask_dev = static_cast<unsigned long long*>(c10::cuda::CUDACachingAllocator::raw_alloc(mask_bytes));

  // dim3 blocks(THCCeilDiv(boxes_num, threadsPerBlock),
  //             THCCeilDiv(boxes_num, threadsPerBlock));
  dim3 blocks(static_cast<unsigned>(col_blocks), static_cast<unsigned>(col_blocks));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads, 0, stream>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  C10_CUDA_CHECK(hipGetLastError());

  // std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  // THCudaCheck(hipMemcpy(&mask_host[0],
  //                       mask_dev,
  //                       sizeof(unsigned long long) * boxes_num * col_blocks,
  //                       hipMemcpyDeviceToHost));

  std::vector<unsigned long long> mask_host(static_cast<size_t>(boxes_num) * col_blocks);
  C10_CUDA_CHECK(hipMemcpy(mask_host.data(), mask_dev, mask_bytes, hipMemcpyDeviceToHost));

  // std::vector<unsigned long long> remv(col_blocks);
  // memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);
  std::vector<unsigned long long> remv(col_blocks, 0ULL);


  at::Tensor keep = at::empty({boxes_num}, boxes.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data_ptr<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  // THCudaFree(state, mask_dev);
  c10::cuda::CUDACachingAllocator::raw_delete(mask_dev);
  // TODO improve this part
  return std::get<0>(order_t.index({
                       keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep).to(
                         order_t.device(), keep.scalar_type())
                     }).sort(0, false));
}
