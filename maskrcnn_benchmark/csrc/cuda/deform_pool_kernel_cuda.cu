#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <algorithm>

using namespace at;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

constexpr int CUDA_NUM_THREADS = 1024;
inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

// ---------------- device helpers ----------------

template <typename T>
__device__ inline T dmin(T a, T b) { return a < b ? a : b; }

template <typename T>
__device__ inline T dmax(T a, T b) { return a > b ? a : b; }

template <typename T>
__device__ inline T dclamp(T x, T lo, T hi) { return dmin(dmax(x, lo), hi); }

// ---------------- bilinear interp ----------------

template <typename scalar_t>
__device__ scalar_t bilinear_interp(
    const scalar_t* data,
    const scalar_t x,
    const scalar_t y,
    const int width,
    const int height) {
  int x1 = floor(x);
  int x2 = ceil(x);
  int y1 = floor(y);
  int y2 = ceil(y);
  scalar_t dist_x = (scalar_t)(x - x1);
  scalar_t dist_y = (scalar_t)(y - y1);
  scalar_t value11 = data[y1 * width + x1];
  scalar_t value12 = data[y2 * width + x1];
  scalar_t value21 = data[y1 * width + x2];
  scalar_t value22 = data[y2 * width + x2];
  scalar_t value =
      (scalar_t)(1 - dist_x) * (scalar_t)(1 - dist_y) * value11 +
      (scalar_t)(1 - dist_x) * dist_y * value12 +
      dist_x * (scalar_t)(1 - dist_y) * value21 +
      dist_x * dist_y * value22;
  return value;
}

// ---------------- forward kernel ----------------

template <typename scalar_t>
__global__ void DeformablePSROIPoolForwardKernel(
    const int count,
    const scalar_t* bottom_data,
    const scalar_t spatial_scale,
    const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const scalar_t* bottom_rois, const scalar_t* bottom_trans,
    const int no_trans,
    const scalar_t trans_std,
    const int sample_per_part,
    const int output_dim,
    const int group_size,
    const int part_size,
    const int num_classes,
    const int channels_each_class,
    scalar_t* top_data,
    scalar_t* top_count) {

  CUDA_KERNEL_LOOP(index, count) {
    // Output order: (n, ctop, ph, pw)
    int pw   = index % pooled_width;
    int ph   = (index / pooled_width) % pooled_height;
    int ctop = (index / pooled_width / pooled_height) % output_dim;
    int n    =  index / pooled_width / pooled_height / output_dim;

    const scalar_t* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = static_cast<int>(offset_bottom_rois[0]);

    scalar_t roi_start_w = (scalar_t)round(offset_bottom_rois[1]) * spatial_scale - (scalar_t)0.5;
    scalar_t roi_start_h = (scalar_t)round(offset_bottom_rois[2]) * spatial_scale - (scalar_t)0.5;
    scalar_t roi_end_w   = ((scalar_t)round(offset_bottom_rois[3]) + (scalar_t)1.) * spatial_scale - (scalar_t)0.5;
    scalar_t roi_end_h   = ((scalar_t)round(offset_bottom_rois[4]) + (scalar_t)1.) * spatial_scale - (scalar_t)0.5;

    // Avoid degenerate ROI
    scalar_t roi_width  = dmax(roi_end_w - roi_start_w, (scalar_t)0.1);
    scalar_t roi_height = dmax(roi_end_h - roi_start_h, (scalar_t)0.1);

    scalar_t bin_size_h = roi_height / (scalar_t)pooled_height;
    scalar_t bin_size_w = roi_width  / (scalar_t)pooled_width;

    scalar_t sub_bin_size_h = bin_size_h / (scalar_t)sample_per_part;
    scalar_t sub_bin_size_w = bin_size_w / (scalar_t)sample_per_part;

    int part_h = floor((scalar_t)ph / (scalar_t)pooled_height * (scalar_t)part_size);
    int part_w = floor((scalar_t)pw / (scalar_t)pooled_width  * (scalar_t)part_size);
    int class_id = ctop / channels_each_class;

    scalar_t trans_x = no_trans ? (scalar_t)0
                                : bottom_trans[(((n * num_classes + class_id) * 2)     * part_size + part_h) * part_size + part_w] * trans_std;
    scalar_t trans_y = no_trans ? (scalar_t)0
                                : bottom_trans[(((n * num_classes + class_id) * 2 + 1) * part_size + part_h) * part_size + part_w] * trans_std;

    scalar_t wstart = (scalar_t)pw * bin_size_w + roi_start_w + trans_x * roi_width;
    scalar_t hstart = (scalar_t)ph * bin_size_h + roi_start_h + trans_y * roi_height;

    scalar_t sum = 0;
    int n_samp = 0;

    int gw = floor((scalar_t)pw * (scalar_t)group_size / (scalar_t)pooled_width);
    int gh = floor((scalar_t)ph * (scalar_t)group_size / (scalar_t)pooled_height);
    gw = dmin(dmax(gw, 0), group_size - 1);
    gh = dmin(dmax(gh, 0), group_size - 1);

    const scalar_t* offset_bottom_data = bottom_data + (roi_batch_ind * channels) * height * width;

    for (int ih = 0; ih < sample_per_part; ih++) {
      for (int iw = 0; iw < sample_per_part; iw++) {
        scalar_t w = wstart + (scalar_t)iw * sub_bin_size_w;
        scalar_t h = hstart + (scalar_t)ih * sub_bin_size_h;

        if (w < (scalar_t)-0.5 || w > (scalar_t)(width - 0.5) ||
            h < (scalar_t)-0.5 || h > (scalar_t)(height - 0.5)) {
          continue;
        }
        w = dclamp(w, (scalar_t)0, (scalar_t)(width  - 1));
        h = dclamp(h, (scalar_t)0, (scalar_t)(height - 1));

        int c = (ctop * group_size + gh) * group_size + gw;
        scalar_t val = bilinear_interp(offset_bottom_data + c * height * width, w, h, width, height);
        sum += val;
        n_samp++;
      }
    }

    top_data[index]  = (n_samp == 0) ? (scalar_t)0 : sum / (scalar_t)n_samp;
    top_count[index] = (scalar_t)n_samp;
  }
}

// ---------------- backward kernel ----------------

template <typename scalar_t>
__global__ void DeformablePSROIPoolBackwardAccKernel(
    const int count,
    const scalar_t* top_diff,
    const scalar_t* top_count,
    const int num_rois,
    const scalar_t spatial_scale,
    const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int output_dim,
    scalar_t* bottom_data_diff, scalar_t* bottom_trans_diff,
    const scalar_t* bottom_data,
    const scalar_t* bottom_rois,
    const scalar_t* bottom_trans,
    const int no_trans,
    const scalar_t trans_std,
    const int sample_per_part,
    const int group_size,
    const int part_size,
    const int num_classes,
    const int channels_each_class) {

  CUDA_KERNEL_LOOP(index, count) {
    int pw   = index % pooled_width;
    int ph   = (index / pooled_width) % pooled_height;
    int ctop = (index / pooled_width / pooled_height) % output_dim;
    int n    =  index / pooled_width / pooled_height / output_dim;

    const scalar_t* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = static_cast<int>(offset_bottom_rois[0]);

    scalar_t roi_start_w = (scalar_t)round(offset_bottom_rois[1]) * spatial_scale - (scalar_t)0.5;
    scalar_t roi_start_h = (scalar_t)round(offset_bottom_rois[2]) * spatial_scale - (scalar_t)0.5;
    scalar_t roi_end_w   = ((scalar_t)round(offset_bottom_rois[3]) + (scalar_t)1.) * spatial_scale - (scalar_t)0.5;
    scalar_t roi_end_h   = ((scalar_t)round(offset_bottom_rois[4]) + (scalar_t)1.) * spatial_scale - (scalar_t)0.5;

    scalar_t roi_width  = dmax(roi_end_w - roi_start_w, (scalar_t)0.1);
    scalar_t roi_height = dmax(roi_end_h - roi_start_h, (scalar_t)0.1);

    scalar_t bin_size_h = roi_height / (scalar_t)pooled_height;
    scalar_t bin_size_w = roi_width  / (scalar_t)pooled_width;

    scalar_t sub_bin_size_h = bin_size_h / (scalar_t)sample_per_part;
    scalar_t sub_bin_size_w = bin_size_w / (scalar_t)sample_per_part;

    int part_h = floor((scalar_t)ph / (scalar_t)pooled_height * (scalar_t)part_size);
    int part_w = floor((scalar_t)pw / (scalar_t)pooled_width  * (scalar_t)part_size);
    int class_id = ctop / channels_each_class;

    scalar_t trans_x = no_trans ? (scalar_t)0
                                : bottom_trans[(((n * num_classes + class_id) * 2)     * part_size + part_h) * part_size + part_w] * trans_std;
    scalar_t trans_y = no_trans ? (scalar_t)0
                                : bottom_trans[(((n * num_classes + class_id) * 2 + 1) * part_size + part_h) * part_size + part_w] * trans_std;

    scalar_t wstart = (scalar_t)pw * bin_size_w + roi_start_w + trans_x * roi_width;
    scalar_t hstart = (scalar_t)ph * bin_size_h + roi_start_h + trans_y * roi_height;

    if (top_count[index] <= (scalar_t)0) {
      continue;
    }

    scalar_t diff_val = top_diff[index] / top_count[index];

    const scalar_t* offset_bottom_data      = bottom_data      + roi_batch_ind * channels * height * width;
    scalar_t*       offset_bottom_data_diff = bottom_data_diff + roi_batch_ind * channels * height * width;

    int gw = floor((scalar_t)pw * (scalar_t)group_size / (scalar_t)pooled_width);
    int gh = floor((scalar_t)ph * (scalar_t)group_size / (scalar_t)pooled_height);
    gw = dmin(dmax(gw, 0), group_size - 1);
    gh = dmin(dmax(gh, 0), group_size - 1);

    for (int ih = 0; ih < sample_per_part; ih++) {
      for (int iw = 0; iw < sample_per_part; iw++) {
        scalar_t w = wstart + (scalar_t)iw * sub_bin_size_w;
        scalar_t h = hstart + (scalar_t)ih * sub_bin_size_h;

        if (w < (scalar_t)-0.5 || w > (scalar_t)(width - 0.5) ||
            h < (scalar_t)-0.5 || h > (scalar_t)(height - 0.5)) {
          continue;
        }
        w = dclamp(w, (scalar_t)0, (scalar_t)(width  - 1));
        h = dclamp(h, (scalar_t)0, (scalar_t)(height - 1));

        int c = (ctop * group_size + gh) * group_size + gw;

        // bilinear weights
        int x0 = floor(w);
        int x1 = ceil(w);
        int y0 = floor(h);
        int y1 = ceil(h);
        scalar_t dist_x = w - (scalar_t)x0;
        scalar_t dist_y = h - (scalar_t)y0;
        scalar_t q00 = (scalar_t)(1 - dist_x) * (scalar_t)(1 - dist_y);
        scalar_t q01 = (scalar_t)(1 - dist_x) * dist_y;
        scalar_t q10 = dist_x * (scalar_t)(1 - dist_y);
        scalar_t q11 = dist_x * dist_y;

        int bottom_index_base = c * height * width;
        atomicAdd(offset_bottom_data_diff + bottom_index_base + y0 * width + x0, q00 * diff_val);
        atomicAdd(offset_bottom_data_diff + bottom_index_base + y1 * width + x0, q01 * diff_val);
        atomicAdd(offset_bottom_data_diff + bottom_index_base + y0 * width + x1, q10 * diff_val);
        atomicAdd(offset_bottom_data_diff + bottom_index_base + y1 * width + x1, q11 * diff_val);

        if (no_trans) {
          continue;
        }

        scalar_t U00 = offset_bottom_data[bottom_index_base + y0 * width + x0];
        scalar_t U01 = offset_bottom_data[bottom_index_base + y1 * width + x0];
        scalar_t U10 = offset_bottom_data[bottom_index_base + y0 * width + x1];
        scalar_t U11 = offset_bottom_data[bottom_index_base + y1 * width + x1];

        scalar_t diff_x =
            (U11 * dist_y + U10 * (scalar_t)(1 - dist_y) -
             U01 * dist_y   - U00 * (scalar_t)(1 - dist_y)) * trans_std * diff_val;
        diff_x *= roi_width;

        scalar_t diff_y =
            (U11 * dist_x + U01 * (scalar_t)(1 - dist_x) -
             U10 * dist_x   - U00 * (scalar_t)(1 - dist_x)) * trans_std * diff_val;
        diff_y *= roi_height;

        atomicAdd(bottom_trans_diff + (((n * num_classes + class_id) * 2)     * part_size + part_h) * part_size + part_w, diff_x);
        atomicAdd(bottom_trans_diff + (((n * num_classes + class_id) * 2 + 1) * part_size + part_h) * part_size + part_w, diff_y);
      }
    }
  }
}

// ---------------- host wrappers ----------------

void DeformablePSROIPoolForward(const at::Tensor data,
                                const at::Tensor bbox,
                                const at::Tensor trans,
                                at::Tensor out,
                                at::Tensor top_count,
                                const int batch,
                                const int channels,
                                const int height,
                                const int width,
                                const int num_bbox,
                                const int channels_trans,
                                const int no_trans,
                                const float spatial_scale,
                                const int output_dim,
                                const int group_size,
                                const int pooled_size,
                                const int part_size,
                                const int sample_per_part,
                                const float trans_std) {
  const int pooled_height = pooled_size;
  const int pooled_width  = pooled_size;
  const int count = num_bbox * output_dim * pooled_height * pooled_width;
  const int num_classes = no_trans ? 1 : channels_trans / 2;
  const int channels_each_class = no_trans ? output_dim : output_dim / num_classes;

  AT_DISPATCH_FLOATING_TYPES(
      data.scalar_type(), "deformable_psroi_pool_forward", ([&] {
        const scalar_t* bottom_data  = data.data_ptr<scalar_t>();
        const scalar_t* bottom_rois  = bbox.data_ptr<scalar_t>();
        const scalar_t* bottom_trans = no_trans ? nullptr : trans.data_ptr<scalar_t>();
        scalar_t*       top_data     = out.data_ptr<scalar_t>();
        scalar_t*       top_count_data = top_count.data_ptr<scalar_t>();

        DeformablePSROIPoolForwardKernel<scalar_t>
            <<<GET_BLOCKS(count), CUDA_NUM_THREADS>>>(
                count, bottom_data, (scalar_t)spatial_scale, channels, height, width,
                pooled_height, pooled_width, bottom_rois, bottom_trans, no_trans,
                (scalar_t)trans_std, sample_per_part, output_dim, group_size,
                part_size, num_classes, channels_each_class, top_data, top_count_data);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in DeformablePSROIPoolForward: %s\n", hipGetErrorString(err));
  }
}

void DeformablePSROIPoolBackwardAcc(const at::Tensor out_grad,
                                    const at::Tensor data,
                                    const at::Tensor bbox,
                                    const at::Tensor trans,
                                    const at::Tensor top_count,
                                    at::Tensor in_grad,
                                    at::Tensor trans_grad,
                                    const int batch,
                                    const int channels,
                                    const int height,
                                    const int width,
                                    const int num_bbox,
                                    const int channels_trans,
                                    const int no_trans,
                                    const float spatial_scale,
                                    const int output_dim,
                                    const int group_size,
                                    const int pooled_size,
                                    const int part_size,
                                    const int sample_per_part,
                                    const float trans_std) {
  const int pooled_height = pooled_size;
  const int pooled_width  = pooled_size;
  const int count = num_bbox * output_dim * pooled_height * pooled_width;
  const int num_classes = no_trans ? 1 : channels_trans / 2;
  const int channels_each_class = no_trans ? output_dim : output_dim / num_classes;

  AT_DISPATCH_FLOATING_TYPES(
      out_grad.scalar_type(), "deformable_psroi_pool_backward_acc", ([&] {
        const scalar_t* top_diff        = out_grad.data_ptr<scalar_t>();
        const scalar_t* bottom_data     = data.data_ptr<scalar_t>();
        const scalar_t* bottom_rois     = bbox.data_ptr<scalar_t>();
        const scalar_t* bottom_trans    = no_trans ? nullptr : trans.data_ptr<scalar_t>();
        scalar_t*       bottom_data_diff  = in_grad.data_ptr<scalar_t>();
        scalar_t*       bottom_trans_diff = no_trans ? nullptr : trans_grad.data_ptr<scalar_t>();
        const scalar_t* top_count_data  = top_count.data_ptr<scalar_t>();

        DeformablePSROIPoolBackwardAccKernel<scalar_t>
            <<<GET_BLOCKS(count), CUDA_NUM_THREADS>>>(
                count, top_diff, top_count_data, num_bbox, (scalar_t)spatial_scale,
                channels, height, width, pooled_height, pooled_width, output_dim,
                bottom_data_diff, bottom_trans_diff, bottom_data, bottom_rois,
                bottom_trans, no_trans, (scalar_t)trans_std, sample_per_part,
                group_size, part_size, num_classes, channels_each_class);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in DeformablePSROIPoolBackwardAcc: %s\n", hipGetErrorString(err));
  }
}
